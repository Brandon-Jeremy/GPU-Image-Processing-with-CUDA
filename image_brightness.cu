
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <png.h>
#include <sys/time.h>
#include <string.h>

typedef struct
{
    int height;
    int width;
    int pixel_size;
    png_infop info_ptr;
    png_byte *buf;
} PNG_RAW;

long long timeInMilliseconds(void)
{
    struct timeval tv;

    gettimeofday(&tv, NULL);
    return (((long long)tv.tv_sec) * 1000) + (tv.tv_usec / 1000);
}

PNG_RAW *read_png(char *file_name)
{
    PNG_RAW *png_raw = (PNG_RAW *)malloc(sizeof(PNG_RAW));

    FILE *fp = fopen(file_name, "rb");
    png_structp png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_read_png(png_ptr, info_ptr, PNG_TRANSFORM_IDENTITY, NULL);
    png_bytepp row_pointers = png_get_rows(png_ptr, info_ptr);
    int width = png_get_image_width(png_ptr, info_ptr);
    int height = png_get_image_height(png_ptr, info_ptr);
    int pixel_size = png_get_rowbytes(png_ptr, info_ptr) / width;
    png_raw->width = width;
    png_raw->height = height;
    png_raw->pixel_size = pixel_size;
    png_raw->buf = (png_byte *)malloc(width * height * pixel_size * sizeof(png_byte));
    png_raw->info_ptr = info_ptr;
    int k = 0;
    for (int i = 0; i < height; i++)
        for (int j = 0; j < width * pixel_size; j++)
        {
            png_raw->buf[k++] = row_pointers[i][j];
        }
    png_destroy_read_struct(&png_ptr, NULL, NULL);
    fclose(fp);
    return png_raw;
}

void write_png(char *file_name, PNG_RAW *png_raw)
{
    FILE *fp = fopen(file_name, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_init_io(png_ptr, fp);
    png_infop info_ptr = png_raw->info_ptr;
    int width = png_raw->width;
    int height = png_raw->height;
    int pixel_size = png_raw->pixel_size;
    png_bytepp row_pointers;
    row_pointers = (png_bytepp)malloc(height * sizeof(png_bytep));
    for (int i = 0; i < height; i++)
        row_pointers[i] = (png_bytep)malloc(width * pixel_size);
    int k = 0;
    for (int i = 0; i < height; i++)
        for (int j = 0; j < width * pixel_size; j++)
        {
            row_pointers[i][j] = png_raw->buf[k++];
        }

    png_set_rows(png_ptr, info_ptr, row_pointers);
    png_write_png(png_ptr, info_ptr, PNG_TRANSFORM_IDENTITY, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    for (int i = 0; i < height; i++)
        free(row_pointers[i]);
    free(row_pointers);
    fclose(fp);
}

__global__ void PictureKernel(png_byte *d_P, int height, int width)
{
    // Calculate the row # of the d_P element
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    // Calculate the column # of the d_P element
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    // each thread computes one element of d_P if in range
    if ((Row < height) && (Col < width))
    {
        d_P[(Row * width + Col) * 3 + 1] = (png_byte)min((d_P[(Row * width + Col) * 3 + 1]*2),255);
        d_P[(Row * width + Col) * 3 + 2] = (png_byte)min((d_P[(Row * width + Col) * 3 + 2]*2),255);
        d_P[(Row * width + Col) * 3 + 3] = (png_byte)min((d_P[(Row * width + Col) * 3 + 3]*2),255);
    }
}

void process_on_device(PNG_RAW *png_raw)
{

    int m = png_raw->height;
    int n = png_raw->width;
    int pixel_size = png_raw->pixel_size;

    dim3 DimGrid((n - 1) / 16 + 1, (m - 1) / 16 + 1, 1);
    dim3 DimBlock(16, 16, 1);

    png_byte *d_P;
    hipError_t err;

    long long start = timeInMilliseconds();

    err = hipMalloc((void **)&d_P, m * n * pixel_size * sizeof(png_byte));
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    hipMemcpy(d_P, png_raw->buf, m * n * pixel_size, hipMemcpyHostToDevice);

    PictureKernel<<<DimGrid, DimBlock>>>(d_P, m, n);

    hipMemcpy(png_raw->buf, d_P, m * n * pixel_size, hipMemcpyDeviceToHost);

    long long end = timeInMilliseconds();

    printf("timing on Device is %lld millis\n", end - start);
}




int main(int argc, char **argv)
{

    PNG_RAW *png_raw = read_png(argv[1]);
    if (png_raw->pixel_size != 3)
    {
        printf("PNG file must have 3 channels not %d\n",png_raw->pixel_size);
        exit(0);
    }
    else
        printf("RGB Processing for Image of %d x %d pixels\n", png_raw->width, png_raw->height);

      process_on_device(png_raw);

    write_png(argv[2], png_raw);

    printf("Processing finished, image converted to brighter img");
}
